#include "hip/hip_runtime.h"
/***********************************************************************************************
 ***                                AAIP       A L G O R I T H M                             ***
 ***********************************************************************************************
 *                                                                                             *
 *                   Paper Name :                                                              *
 *                                                                                             *
 *                   Authors :  Shunran ZHANG, Xiubo ZHANG                                     *
 *                                                                                             *
 *                   File Name : emd_cuda.cu                                                   *
 *                                                                                             *
 *                   Programmer : Xiubo ZHANG                                                  *
 *                                                                                             *
 *                   Start Date : Mar 3, 2023                                                  *
 *                                                                                             *
 *                   Last Update : July 27, 2023                                              *
 *                                                                                             *
 *---------------------------------------------------------------------------------------------*/
#include <stdio.h>
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>

__device__ __forceinline__ float atomicMax(float *address, float val)
{
    int ret = __float_as_int(*address);
    while(val > __int_as_float(ret))
    {
        int old = ret;
        if((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
            break;
    }
    return __int_as_float(ret);
}


__global__ void clear_here(int b, int * cnt_tmp, int * unass_cnt) {
	for (int i = threadIdx.x; i < b; i += blockDim.x) {
		cnt_tmp[i] = 0;
		unass_cnt[i] = 0;
	}
}

__global__ void calc_unass_cnt_here(int b, int n, int * assignment, int * unass_cnt) { 
	// count the number of unassigned points in each batch
	const int BLOCK_SIZE = 1024; 
	__shared__ int scan_array[BLOCK_SIZE];
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		scan_array[threadIdx.x] = assignment[i * n + blockIdx.y * BLOCK_SIZE + threadIdx.x] == -1 ? 1 : 0;
		__syncthreads();
		
		int stride = 1;
		while(stride <= BLOCK_SIZE / 2) {
			int index = (threadIdx.x + 1) * stride * 2 - 1; 
			if(index < BLOCK_SIZE)
				scan_array[index] += scan_array[index - stride]; 
			stride = stride * 2;
			__syncthreads(); 
		}
		__syncthreads();
		
		if (threadIdx.x == BLOCK_SIZE - 1) {
			atomicAdd(&unass_cnt[i], scan_array[threadIdx.x]);
		}
		__syncthreads();
	}
}

__global__ void calc_unass_cnt_sum_here(int b, int * unass_cnt, int * unass_cnt_sum) {
	// count the cumulative sum over over unass_cnt
	const int BLOCK_SIZE = 512; // batch_size <= 512
	__shared__ int scan_array[BLOCK_SIZE];
	scan_array[threadIdx.x] = unass_cnt[threadIdx.x];
	__syncthreads();
	
	int stride = 1;
	while(stride <= BLOCK_SIZE / 2) {
		int index = (threadIdx.x + 1) * stride * 2 - 1; 
		if(index < BLOCK_SIZE)
			scan_array[index] += scan_array[index - stride]; 
		stride = stride * 2;
		__syncthreads(); 
	}
	__syncthreads();
	stride = BLOCK_SIZE / 4; 
	while(stride > 0) {
		int index = (threadIdx.x + 1) * stride * 2 - 1; 
		if((index + stride) < BLOCK_SIZE)
			scan_array[index + stride] += scan_array[index];
		stride = stride / 2;
		__syncthreads(); 
	}
	__syncthreads(); 
	
	//printf("%d\n", unass_cnt_sum[b - 1]);
	unass_cnt_sum[threadIdx.x] = scan_array[threadIdx.x];
}

__global__ void calc_unass_idx_here(int b, int n, int * assignment, int * unass_idx, int * unass_cnt, int * unass_cnt_sum, int * cnt_tmp) {
	// list all the unassigned points
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		if (assignment[i * n + blockIdx.y * 1024 + threadIdx.x] == -1) {
			int idx = atomicAdd(&cnt_tmp[i], 1);
			unass_idx[unass_cnt_sum[i] - unass_cnt[i] + idx] = blockIdx.y * 1024 + threadIdx.x;
		} 
	}
}

__global__ void Bid_here(int b, int n, const float * xyz1, const float * xyz2, float eps, int * assignment, int * assignment_inv, float * price, 
					int * bid, float * bid_increments, float * max_increments, int * unass_cnt, int * unass_cnt_sum, int * unass_idx) {
	const int batch = 2048, block_size = 1024, block_cnt = n / 1024;
	__shared__ float xyz2_buf[batch * 3];
	__shared__ float price_buf[batch];
	__shared__ float best_buf[block_size];
	__shared__ float better_buf[block_size];
	__shared__ int best_i_buf[block_size];
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		int _unass_cnt = unass_cnt[i];
		if (_unass_cnt == 0)
			continue;
		int _unass_cnt_sum = unass_cnt_sum[i];
		int unass_per_block = (_unass_cnt + block_cnt - 1) / block_cnt;
		int thread_per_unass = block_size / unass_per_block;
		int unass_this_block = max(min(_unass_cnt - (int) blockIdx.y * unass_per_block, unass_per_block), 0);
			
		float x1, y1, z1, best = -1e9, better = -1e9;
		int best_i = -1, _unass_id = -1, thread_in_unass;

		if (threadIdx.x < thread_per_unass * unass_this_block) {
			_unass_id = unass_per_block * blockIdx.y + threadIdx.x / thread_per_unass + _unass_cnt_sum - _unass_cnt;
			_unass_id = unass_idx[_unass_id];
			thread_in_unass = threadIdx.x % thread_per_unass;

			x1 = xyz1[(i * n + _unass_id) * 3 + 0];
			y1 = xyz1[(i * n + _unass_id) * 3 + 1];
			z1 = xyz1[(i * n + _unass_id) * 3 + 2];
		}

		for (int k2 = 0; k2 < n; k2 += batch) {
			int end_k = min(n, k2 + batch) - k2;
			for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
				xyz2_buf[j] = xyz2[(i * n + k2) * 3 + j];
			}
			for (int j = threadIdx.x; j < end_k; j += blockDim.x) {
				price_buf[j] = price[i * n + k2 + j];
			}
			__syncthreads();

			if (_unass_id != -1) {
				int delta = (end_k + thread_per_unass - 1) / thread_per_unass;
				int l = thread_in_unass * delta;
				int r = min((thread_in_unass + 1) * delta, end_k);
				for (int k = l; k < r; k++) 
				//if (!last || assignment_inv[i * n + k + k2] == -1)
				{
					float x2 = xyz2_buf[k * 3 + 0] - x1;
					float y2 = xyz2_buf[k * 3 + 1] - y1;
					float z2 = xyz2_buf[k * 3 + 2] - z1;
					// the coordinates of points should be normalized to [0, 1]
					float d = 3.0 - sqrtf(x2 * x2 + y2 * y2 + z2 * z2) - price_buf[k];
					if (d > best) {
						better = best;
						best = d;
						best_i = k + k2;
					}
					else if (d > better) {
						better = d;
					}
				}
			}
			__syncthreads();
		}

		best_buf[threadIdx.x] = best;
		better_buf[threadIdx.x] = better;
		best_i_buf[threadIdx.x] = best_i;
		__syncthreads();
		
		if (_unass_id != -1 && thread_in_unass == 0) {
			for (int j = threadIdx.x + 1; j < threadIdx.x + thread_per_unass; j++) {
				if (best_buf[j] > best) {
					better = max(best, better_buf[j]);
					best = best_buf[j];
					best_i = best_i_buf[j];
				}
				else better = max(better, best_buf[j]);
			}
			bid[i * n + _unass_id] = best_i;
			bid_increments[i * n + _unass_id] = best - better + eps; 
			atomicMax(&max_increments[i * n + best_i], best - better + eps);
		}
	}
}





int cuda_auction_find_conflict(at::Tensor xyz1, at::Tensor xyz2, at::Tensor dist, at::Tensor assignment, at::Tensor price, 
	                 at::Tensor assignment_inv, at::Tensor bid, at::Tensor bid_increments, at::Tensor max_increments,
	                 at::Tensor unass_idx, at::Tensor unass_cnt, at::Tensor unass_cnt_sum, at::Tensor cnt_tmp, at::Tensor max_idx,at::Tensor is_conflict, float eps, int iters) {

	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B
	
	if (n != m) {
		printf("Input Error! The two point clouds should have the same size.\n");
		return -1;
	}

	if (batch_size > 512) {
		printf("Input Error! The batch size should be less than 512.\n");
		return -1;
	}

	if (n % 1024 != 0) {
		printf("Input Error! The size of the point clouds should be a multiple of 1024.\n");
		return -1;
	}

	//hipEvent_t start,stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//hipEventRecord(start);
	//int iters = 50;
	// SIA_PRE_ASSIGN<<<dim3(batch_size, n / 1024, 1), 1024>>>(batch_size, n, xyz1.data<float>(), xyz2.data<float>(), eps, assignment.data<int>(), assignment_inv.data<int>(), 
	// 		                          price.data<float>(), bid.data<int>(), bid_increments.data<float>(), max_increments.data<float>(),
	// 		                          unass_cnt.data<int>(), unass_cnt_sum.data<int>(), unass_idx.data<int>());
        
		clear_here<<<1, batch_size>>>(batch_size, cnt_tmp.data<int>(), unass_cnt.data<int>());
		calc_unass_cnt_here<<<dim3(batch_size, n / 1024, 1), 1024>>>(batch_size, n, assignment.data<int>(), unass_cnt.data<int>());
		calc_unass_cnt_sum_here<<<1, batch_size>>>(batch_size, unass_cnt.data<int>(), unass_cnt_sum.data<int>());
		calc_unass_idx_here<<<dim3(batch_size, n / 1024, 1), 1024>>>(batch_size, n, assignment.data<int>(), unass_idx.data<int>(), unass_cnt.data<int>(), 
											 unass_cnt_sum.data<int>(), cnt_tmp.data<int>());
		Bid_here<<<dim3(batch_size, n / 1024, 1), 1024>>>(batch_size, n, xyz1.data<float>(), xyz2.data<float>(), eps, assignment.data<int>(), assignment_inv.data<int>(), 
			                          price.data<float>(), bid.data<int>(), bid_increments.data<float>(), max_increments.data<float>(),
			                          unass_cnt.data<int>(), unass_cnt_sum.data<int>(), unass_idx.data<int>());
		
		
	

    
	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd Output: %s\n", hipGetErrorString(err));
	    return 0;
	  }
	  return 1;
}





